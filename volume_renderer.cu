#include "hip/hip_runtime.h"
// This file is distributed under the MIT license.
// See the LICENSE file for details.

#include <cassert>

#include <thrust/copy.h>

#include <visionaray/detail/platform.h>

#include <visionaray/math/math.h>

#include <vkt/InputStream.hpp>
#include <vkt/LookupTable.hpp>
#include <vkt/RawFile.hpp>
#include <vkt/StructuredVolume.hpp>

#include <common/timer.h>

#include "volume_renderer.h"

namespace visionaray
{

volume_renderer::volume_renderer(std::string filename)
    : filename(filename)
{
    vkt::RawFile file(filename.c_str(), "r");
    vkt::Vec3i dims = file.getDims();
    uint16_t bpv = file.getBytesPerVoxel();

    volume = vkt::StructuredVolume(dims.x, dims.y, dims.z, bpv);
    vkt::InputStream is(file);
    is.read(volume);

    float rgba[] = {
        //1.f, 1.f, 1.f, .005f,
        //1.f, 1.f, 1.f, 0.f,
        0.f, 0.f, .2f, .005f,
        0.f, .1f, .1f, .25f,
        .5f, .5f, .7f, .5f,
        .7f, .7f, .07f, .75f,
        1.f, .3f, .3f, 1.f
    };
    lut = vkt::LookupTable(5,1,1,vkt::ColorFormat::RGBA32F);
    lut.setData((uint8_t*)rgba);

    vkt::Vec3f dist = volume.getDist();
    bbox = aabb(
            { 0.f, 0.f, 0.f },
            { dims.x * dist.x, dims.y * dist.y, dims.z * dist.z }
            );

    device_volume = cuda_texture<uint8_t, 3>((size_t)dims.x, (size_t)dims.y, (size_t)dims.z);
    device_volume.reset(volume.getData());
    device_volume.set_filter_mode(Nearest);
    device_volume.set_address_mode(Clamp);

    device_transfunc = cuda_texture<vec4, 1>((size_t)lut.getDims().x);
    device_transfunc.reset((vec4*)lut.getData());
    device_transfunc.set_filter_mode(Linear);
    device_transfunc.set_address_mode(Clamp);
}

void volume_renderer::resize(int w, int h)
{
    std::cout << w << ' ' << h << '\n';
    device_rt.resize(w, h);

    device_spheres.resize(w * h);
    device_sphere_colors.resize(w * h);

    host_spheres.resize(w * h);
    host_sphere_colors.resize(w * h);
}

// Stolen from Ingo's owl
/*! simple 24-bit linear congruence generator */
template<unsigned int N=4>
struct LCG {

  inline __device__ LCG()
  { /* intentionally empty so we can use it in device vars that
       don't allow dynamic initialization (ie, PRD) */
  }
  inline __device__ LCG(unsigned int val0, unsigned int val1)
  { init(val0,val1); }

  inline __device__ void init(unsigned int val0, unsigned int val1)
  {
    unsigned int v0 = val0;
    unsigned int v1 = val1;
    unsigned int s0 = 0;

    for (unsigned int n = 0; n < N; n++) {
      s0 += 0x9e3779b9;
      v0 += ((v1<<4)+0xa341316c)^(v1+s0)^((v1>>5)+0xc8013ea4);
      v1 += ((v0<<4)+0xad90777d)^(v0+s0)^((v0>>5)+0x7e95761e);
    }
    state = v0;
  }

  // Generate random unsigned int in [0, 2^24)
  inline __device__ float operator() ()
  {
    const uint32_t LCG_A = 1664525u;
    const uint32_t LCG_C = 1013904223u;
    state = (LCG_A * state + LCG_C);
    return ldexpf(float(state), -32);
    // return (state & 0x00FFFFFF) / (float) 0x01000000;
  }

  uint32_t state;
};

struct kernel
{
    using S = float;

    __device__
    result_record<S> operator()(basic_ray<float> ray, int x, int y)
    {
        LCG<4> rnd(x,y);
        result_record<S> result;

        auto hit_rec = intersect(ray, bbox);
        auto t = hit_rec.tnear;

        result.color = vec4(0.0);

        spheres[y * width + x] = vec4(0.f);
        sphere_colors[y * width + x] = vec4(0.f);

    
        int it = 0;
        vec4 oldColor(0.f);
        bool dont = false;
        while (t < hit_rec.tfar)
        {
            auto pos = ray.ori + ray.dir * t;
            auto tex_coord = pos / vector<3, S>(bbox.size());

            // sample volume and do post-classification
            float voxel = convert_to_float(tex3D(volume_ref, tex_coord)) / S(255.f);
            vec4 color = tex1D(transfunc_ref, voxel);

            // opacity correction
            color.w = S(1.f) - pow(S(1.f) - color.w, S(dt));

            // premultiplied alpha
            color.xyz() *= color.w;

            // front-to-back alpha compositing
            result.color += select(
                    t < hit_rec.tfar,
                    color * (1.0f - result.color.w),
                    vec4(0.0)
                    );

            //if (!dont && (it == 0 || color.w > 0.1f))
            if (!dont && result.color.w > 0.8f)
            {
                spheres[y * width + x] = vec4(pos, .5f);
                dont = true;
            }

            ++it;

            // early-ray termination - don't traverse w/o a contribution
            if ( all(result.color.w >= 0.999) )
            {
                break;
            }

            // step on
            t += dt;
            oldColor = color;
        }

        if (!dont && result.color.w > 1e-8f)
        {
            auto pos = ray.ori + ray.dir * hit_rec.tfar;
            //auto pos = ray.ori + ray.dir * lerp(hit_rec.tnear,hit_rec.tfar,rnd());
            spheres[y * width + x] = vec4(pos, .5f);
        }

        sphere_colors[y * width + x] = result.color;

        result.hit = hit_rec.hit;
        return result;
    }

    vec4* spheres;
    vec4* sphere_colors;
    cuda_texture_ref<uint8_t, 3> volume_ref;
    cuda_texture_ref<vec4, 1> transfunc_ref;
    aabb bbox;
    int width;
    float dt;
};

void volume_renderer::render(pinhole_camera const& cam)
{
    // some setup

    using R = basic_ray<float>;
    using S = R::scalar_type;
    using C = vector<4, S>;

    auto sparams = make_sched_params(cam, device_rt);

    // call kernel in schedulers' frame() method

    kernel kern;
    kern.spheres = thrust::raw_pointer_cast(device_spheres.data());
    kern.sphere_colors = thrust::raw_pointer_cast(device_sphere_colors.data());
    kern.volume_ref = cuda_texture_ref<uint8_t, 3>(device_volume);
    kern.transfunc_ref = cuda_texture_ref<vec4, 1>(device_transfunc);
    kern.bbox = bbox;
    kern.width = device_rt.width();
    kern.dt = .5f;

    cuda::timer t;
    device_sched.frame(kern, sparams);
    std::cout << "Rendered. Elapsed: " << t.elapsed() << '\n';

    thrust::copy(device_spheres.begin(), device_spheres.end(), host_spheres.begin());
    thrust::copy(device_sphere_colors.begin(), device_sphere_colors.end(), host_sphere_colors.begin());
}

vec4 const* volume_renderer::color_buffer() const
{
    return host_sphere_colors.data();
}

vec4 const* volume_renderer::object_space_samples() const
{
    return host_spheres.data();
}

size_t volume_renderer::num_samples() const
{
    assert(host_sphere_colors.size() == host_spheres.size());

    return host_spheres.size();
}

} // visionaray
