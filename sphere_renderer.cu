#include "hip/hip_runtime.h"
// This file is distributed under the MIT license.
// See the LICENSE file for details.

#include <thrust/device_vector.h>

#include <visionaray/aligned_vector.h>
#include <visionaray/bvh.h>
#include <visionaray/pinhole_camera.h>
#include <visionaray/pixel_unpack_buffer_rt.h>
#include <visionaray/result_record.h>
#include <visionaray/scheduler.h>
#include <visionaray/traverse.h>

#include <common/timer.h>

#include "sphere_renderer.h"

namespace visionaray
{

//-------------------------------------------------------------------------------------------------
// Solid sphere, one that a ray can travel through from tnear to tfar
//

struct solid_sphere : basic_sphere<float>
{
};

template <typename S>
struct solid_hit_record : visionaray::hit_record<basic_ray<S>, primitive<unsigned>>
{
    S tfar;
    S radius2;
};

template <typename S, typename Cond>
__device__
void update_if(solid_hit_record<S>& dst, solid_hit_record<S> const& src, Cond const& cond)
{
    dst.hit        |= cond;
    dst.t           = select( cond, src.t, dst.t );
    dst.prim_id     = select( cond, src.prim_id, dst.prim_id );
    dst.geom_id     = select( cond, src.geom_id, dst.geom_id );
    dst.u           = select( cond, src.u, dst.u );
    dst.v           = select( cond, src.v, dst.v );

    dst.tfar        = select( cond, src.tfar , dst.tfar  );
    dst.radius2     = select( cond, src.radius2 , dst.radius2  );
}

void split_primitive(aabb& L, aabb& R, float plane, int axis, solid_sphere const& prim)
{
    split_primitive(L, R, plane, axis, static_cast<basic_sphere<float>>(prim));
}


template <typename S>
__device__
solid_hit_record<S> intersect(
        basic_ray<S> const& ray,
        solid_sphere const& sphere
        )
{
    typedef basic_ray<S> ray_type;
    typedef vector<3, S> vec_type;

    ray_type r = ray;
    r.ori -= vec_type( sphere.center );

    auto A = dot(r.dir, r.dir);
    auto B = dot(r.dir, r.ori) * S(2.0);
    auto C = dot(r.ori, r.ori) - sphere.radius * sphere.radius;

    // solve Ax**2 + Bx + C
    auto disc = B * B - S(4.0) * A * C;
    auto valid = disc >= S(0.0);

    auto root_disc = select(valid, sqrt(disc), disc);

    auto q = select( B < S(0.0), S(-0.5) * (B - root_disc), S(-0.5) * (B + root_disc) );

    auto tnear = q / A;
    auto tfar = C / q;

    auto mask = tnear > tfar;
    auto tmp = select(mask, tnear, S(0.0));
    tnear = select(mask, tfar, tnear);
    tfar = select(mask, tmp, tfar);

    valid &= tnear > S(0.0);

    solid_hit_record<S> result;
    result.hit = valid;
    result.prim_id = sphere.prim_id;
    result.geom_id = sphere.geom_id;
    result.t = select( valid, tnear, S(-1.0) );
    result.tfar = select( valid, tfar, S(-1.0) );
    result.radius2 = select( valid, sphere.radius + sphere.radius, S(-1.0) );
    return result;
}


struct render_kernel
{
    __device__
    result_record<float> operator()(basic_ray<float> ray, int x, int y)
    {
        result_record<float> result;
        result.color = vec4(0.0);

#ifdef __CUDA_ARCH__
        // Perform multi-hit, we allow for up to 16 hits
        // Multi-hit returns a sorted array (based on
        // ray parameter "t") of hit records
        auto hit_rec = closest_hit(ray, bvhs, bvhs + 1);

        // Use closest hit in the sequence
        // for visibility testing
        result.hit = hit_rec.hit;
        if (hit_rec.hit)
        {
            result.color = sphere_colors[hit_rec.prim_id];
            result.isect_pos  = ray.ori + ray.dir * hit_rec.t;
        }
#endif

        return result;
    }

    cuda_index_bvh<solid_sphere>::bvh_ref* bvhs;
    vec4* sphere_colors;
};


struct sphere_renderer::impl
{
    thrust::device_vector<vec4> device_sphere_colors;
    pixel_unpack_buffer_rt<PF_RGBA8, PF_UNSPECIFIED> host_rt;
    cuda_sched<basic_ray<float>> host_sched;
    cuda_index_bvh<solid_sphere> host_bvh;
};

sphere_renderer::sphere_renderer()
    : impl_(new impl)
{
}

sphere_renderer::~sphere_renderer()
{
}

void sphere_renderer::reset(const vec4* spheres, const vec4* colors, size_t num_spheres)
{
    std::vector<vec4> sphere_colors;

    std::vector<solid_sphere> sphere_geom;

    unsigned prim_id = 0;
    for (size_t i = 0; i < num_spheres; ++i)
    {
        if (spheres[i].w > 0.f)
        {
            solid_sphere sp;
            sp.center = spheres[i].xyz();
            sp.radius = spheres[i].w;
            sphere_geom.push_back(sp);
            sphere_geom.back().prim_id = prim_id++;
            sphere_colors.push_back(colors[i]);
        }
    }

    thrust::device_vector<solid_sphere> d_sphere_geom(sphere_geom);
    impl_->device_sphere_colors.resize(sphere_colors.size());
    thrust::copy(sphere_colors.begin(), sphere_colors.end(), impl_->device_sphere_colors.begin());

    lbvh_builder builder;

    cuda::timer t;
    impl_->host_bvh = builder.build(cuda_index_bvh<solid_sphere>{},
                                    thrust::raw_pointer_cast(d_sphere_geom.data()),
                                    d_sphere_geom.size());
    std::cout << t.elapsed() << '\n';
}

void sphere_renderer::resize(int w, int h)
{
    impl_->host_rt.resize(w, h);
}

void sphere_renderer::render(pinhole_camera const& cam)
{
    if (impl_->device_sphere_colors.empty())
        return;

    // some setup

    auto sparams = make_sched_params(
            cam,
            impl_->host_rt
            );

    using bvh_ref = cuda_index_bvh<solid_sphere>::bvh_ref;

    thrust::device_vector<bvh_ref> bvhs;
    bvhs.push_back(impl_->host_bvh.ref());

    render_kernel kern;
    kern.bvhs = thrust::raw_pointer_cast(bvhs.data());
    kern.sphere_colors = thrust::raw_pointer_cast(impl_->device_sphere_colors.data());

    impl_->host_sched.frame(kern, sparams);

    // display the rendered image
    impl_->host_rt.display_color_buffer();
}

} // visionaray
